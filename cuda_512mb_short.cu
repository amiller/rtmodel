/* Andrew Miller <amiller@dappervision.com>
 *
 * Cuda 512*512*512*4bytes test
 * 
 * According to the KinectFusion UIST 2011 paper, it's possible 
 * to do a sweep of 512^3 voxels, 32-bits each, in ~2ms on a GTX470.
 * 
 * This code is a simple benchmark accessing 512^3*2 short ints.
 * voxel has two 16-bit components. In this benchmark kernel, we
 * simply increment these values by a constant K. More than anything
 * it's a test of the memory bandwidth.
 *
 * On my GTX470 card, this kernel takes 10.7ms instead of ~2ms. Is there
 * a faster way to do this?
 *
 * Citation: http://dl.acm.org/citation.cfm?id=2047270 
 * Public gdocs pdf link: http://tinyurl.com/6xlznbx
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <assert.h>

const int N_DATA = (512*512*512*2);
const int N_BYTES = (N_DATA*2);
const int N_GRID = 512;
const int N_BLOCK = 512;
const int N_CHUNK = 8;
const int N_FAN = N_DATA/N_GRID/N_BLOCK/N_CHUNK;
const int K = 13;
const int N_LOOPS = 10;

/*
  Each kernel processes several adjacent elements
  N_DATA = (N_GRID) * (N_FAN) * (N_BLOCK) * (N_CHUNK) = 512*512*512*2
 */

__global__ void incr_data1(short int *data) {
  // Outer loop skips by strides of N_BLOCK*N_CHUNK
  for (int i = 0; i < N_FAN; i++) {
    int idx = blockIdx.x*(N_FAN*N_BLOCK*N_CHUNK) + i*(N_BLOCK*N_CHUNK) + threadIdx.x*(N_CHUNK);

    // Inner loop processes 16 bytes (8 short ints) at once (a chunk)
    #pragma unroll
    for (int j = 0; j < N_CHUNK; j++, idx++) {
      data[idx] += K;
    }
  }
}

int main(void) {
  short int *data_gpu;
  short int *data_cpu;
  
  hipMalloc((void **) &data_gpu, N_BYTES);
  data_cpu = (short int *) calloc(N_BYTES, 1);
  hipMemcpy(data_gpu, data_cpu, N_BYTES, hipMemcpyHostToDevice);

  dim3 dimBlock(N_BLOCK,1,1);
  dim3 dimGrid(N_GRID,1,1);    

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start);

  // Run the kernel several times
  for (int i = 0; i < N_LOOPS; i++) {      
    incr_data1<<<dimGrid, dimBlock>>>(data_gpu);
  }

  hipEventRecord(e_stop);
  hipEventSynchronize(e_stop);

  // Copy back to the host and check we have what we expect
  hipMemcpy(data_cpu, data_gpu, N_BYTES, hipMemcpyDeviceToHost);
  for (int i = 0; i < N_DATA; i++) {
    assert(data_cpu[i] == (short)N_LOOPS*K);
  }

  // Timing information
  float ms;
  hipEventElapsedTime(&ms, e_start, e_stop);  
  printf("%d sweeps of %.1f megabytes in %.1fms (avg %.1fms)\n", 
	 N_LOOPS, N_BYTES/1000.0/1000.0, ms, ms/N_LOOPS);

  hipFree(data_gpu);
  free(data_cpu);
  return 0;
}
